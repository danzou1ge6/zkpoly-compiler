#include "poly.h"
#include "poly_basic.cuh"
#include "poly_eval.cuh"
#include "kate_division.cuh"
#include "scan_mul.cuh"

hipError_t poly_add(unsigned int *result, const unsigned int *a, const unsigned int *b, unsigned long long len, hipStream_t stream) {
    unsigned int block = 256;
    unsigned int grid = (len - 1) / block + 1;
    detail::poly_add<POLY_FIELD><<<grid, block, 0, stream>>>(a, b, result, len);
    CUDA_CHECK(hipGetLastError());
    return hipSuccess;
}

hipError_t poly_sub(unsigned int *result, const unsigned int *a, const unsigned int *b, unsigned long long len, hipStream_t stream) {
    unsigned int block = 256;
    unsigned int grid = (len - 1) / block + 1;
    detail::poly_sub<POLY_FIELD><<<grid, block, 0, stream>>>(a, b, result, len);
    CUDA_CHECK(hipGetLastError());
    return hipSuccess;
}

hipError_t poly_mul(unsigned int *result, const unsigned int *a, const unsigned int *b, unsigned long long len, hipStream_t stream) {
    unsigned int block = 256;
    unsigned int grid = (len - 1) / block + 1;
    detail::poly_mul<POLY_FIELD><<<grid, block, 0, stream>>>(a, b, result, len);
    CUDA_CHECK(hipGetLastError());
    return hipSuccess;
}

hipError_t poly_one(unsigned int * target, unsigned long long len, hipStream_t stream) {
    return detail::poly_one<POLY_FIELD>(target, len, stream);
}

hipError_t poly_zero(unsigned int * target, unsigned long long len, hipStream_t stream) {
    return detail::poly_zero<POLY_FIELD>(target, len, stream);
}

hipError_t poly_eval(void* temp_buf, unsigned long *temp_buf_size, const unsigned int *poly,  unsigned int* res, const unsigned int*x, unsigned long long len, hipStream_t stream) {
    return detail::poly_eval<POLY_FIELD>(temp_buf, temp_buf_size, poly, res, reinterpret_cast<const POLY_FIELD*>(x), len, stream);
}

hipError_t kate_division(void* temp_buf, unsigned long *temp_buf_size, unsigned int log_p, const unsigned int *p, const unsigned int *b, unsigned int *q, hipStream_t stream) {
    return detail::kate_division<POLY_FIELD>(temp_buf, temp_buf_size, log_p, reinterpret_cast<const POLY_FIELD*>(p), reinterpret_cast<const POLY_FIELD*>(b), reinterpret_cast<POLY_FIELD*>(q), stream);
}

hipError_t scan_mul(void * temp_buffer, unsigned long *buffer_size, const unsigned int *poly, unsigned int *target, const unsigned int *x0, unsigned long long len, hipStream_t stream) {
    return detail::scan_mul<POLY_FIELD>(temp_buffer, buffer_size, poly, target, x0, len, stream);
}
