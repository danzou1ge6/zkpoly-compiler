#include "hip/hip_runtime.h"
#include "poly.h"
#include "poly_basic.cuh"
#include "poly_eval.cuh"
#include "kate_division.cuh"
#include "scan_mul.cuh"
#include "batched_invert.cuh"

hipError_t poly_add(PolyPtr r, ConstPolyPtr a, ConstPolyPtr b, hipStream_t stream) {
    auto len = std::min(a.len, b.len);
    unsigned int block = 256;
    unsigned int grid = (len - 1) / block + 1;
    auto a_iter = iter::make_slice_iter<POLY_FIELD>(a);
    auto b_iter = iter::make_slice_iter<POLY_FIELD>(b);
    auto r_iter = iter::make_slice_iter<POLY_FIELD>(r);
    detail::poly_add<POLY_FIELD><<<grid, block, 0, stream>>>(a_iter, b_iter, r_iter, len);
    CUDA_CHECK(hipGetLastError());
    return hipSuccess;
}

hipError_t poly_sub(PolyPtr r, ConstPolyPtr a, ConstPolyPtr b, hipStream_t stream) {
    auto len = std::min(a.len, b.len);
    unsigned int block = 256;
    unsigned int grid = (len - 1) / block + 1;
    auto a_iter = iter::make_slice_iter<POLY_FIELD>(a);
    auto b_iter = iter::make_slice_iter<POLY_FIELD>(b);
    auto r_iter = iter::make_slice_iter<POLY_FIELD>(r);
    detail::poly_sub<POLY_FIELD><<<grid, block, 0, stream>>>(a_iter, b_iter, r_iter, len);
    CUDA_CHECK(hipGetLastError());
    return hipSuccess;
}

hipError_t poly_mul(PolyPtr r, ConstPolyPtr a, ConstPolyPtr b, hipStream_t stream) {
    auto len = std::min(a.len, b.len);
    unsigned int block = 256;
    unsigned int grid = (len - 1) / block + 1;
    auto a_iter = iter::make_slice_iter<POLY_FIELD>(a);
    auto b_iter = iter::make_slice_iter<POLY_FIELD>(b);
    auto r_iter = iter::make_slice_iter<POLY_FIELD>(r);
    detail::poly_mul<POLY_FIELD><<<grid, block, 0, stream>>>(a_iter, b_iter, r_iter, len);
    CUDA_CHECK(hipGetLastError());
    return hipSuccess;
}

hipError_t poly_one_lagrange(PolyPtr target, hipStream_t stream) {
    return detail::poly_one_lagrange<POLY_FIELD>(target, stream);
}

hipError_t poly_one_coef(PolyPtr target, hipStream_t stream) {
    return detail::poly_one_coef<POLY_FIELD>(target, stream);
}

hipError_t poly_zero(PolyPtr target, hipStream_t stream) {
    return detail::poly_zero<POLY_FIELD>(target, stream);
}

hipError_t poly_eval(void* temp_buf, unsigned long *temp_buf_size, ConstPolyPtr poly,  unsigned int* res, const unsigned int*x, hipStream_t stream) {
    return detail::poly_eval<POLY_FIELD>(temp_buf, temp_buf_size, poly, res, reinterpret_cast<const POLY_FIELD*>(x), stream);
}

hipError_t kate_division(void* temp_buf, unsigned long *temp_buf_size, unsigned int log_p, ConstPolyPtr p, const unsigned int *b, PolyPtr q, hipStream_t stream) {
    return detail::kate_division<POLY_FIELD>(temp_buf, temp_buf_size, log_p, p, reinterpret_cast<const POLY_FIELD*>(b), q, stream);
}

hipError_t scan_mul(void * temp_buffer, unsigned long *buffer_size, PolyPtr target, hipStream_t stream) {
    return detail::scan_mul<POLY_FIELD>(temp_buffer, buffer_size, target, stream);
}

hipError_t batched_invert(void *temp_buffer, unsigned long *buffer_size, PolyPtr poly, unsigned int *inv, hipStream_t stream) {
    return detail::batched_invert<POLY_FIELD>(temp_buffer, buffer_size, poly, inv, stream);
}

hipError_t inv_scalar(unsigned int* target, hipStream_t stream) {
    detail::inverse_scalar<POLY_FIELD><<< 1, 1, 0, stream >>>(reinterpret_cast<POLY_FIELD*>(target));
    return hipGetLastError();
}

hipError_t scalar_pow(unsigned int* target, unsigned long long exp, hipStream_t stream) {
    detail::scalar_pow<POLY_FIELD><<< 1, 1, 0, stream >>>(reinterpret_cast<POLY_FIELD*>(target), exp);
    return hipGetLastError();
}