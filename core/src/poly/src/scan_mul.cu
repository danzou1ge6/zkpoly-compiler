#include "scan_mul.cuh"

hipError_t scan_mul(void * temp_buffer, unsigned long *buffer_size, PolyPtr target, hipStream_t stream) {
    return detail::scan_mul<POLY_FIELD>(temp_buffer, buffer_size, target, stream);
}