#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest/doctest.h>
#include "../src/batched_invert.cuh"
#include <iostream>
#include <chrono>

using mont::u32;
using mont::u64;
using mont::usize;
using Field = bn254_fr::Element;
using Number = mont::Number<Field::LIMBS>;

TEST_CASE("gpu invert") {
    std::cout << "testing the gpu invert" << std::endl;
    Field * p, *q;
    u32 log_len = 10;
    u32 len = 1 << log_len;
    p = new Field [len];
    q = new Field [len];

    for (u64 i = 0; i < len; i++) {
        if (i % 7 == 0) {
            p[i] = Field::zero();
        } else {
            p[i] = Field::host_random();
        }
    }

    Field *p_d;
    hipMalloc(&p_d, len * sizeof(Field));
    hipMemcpy(p_d, p, len * sizeof(Field), hipMemcpyHostToDevice);

    void *temp_buffer;
    usize buffer_size = 0;
    auto p_ptr = PolyPtr{reinterpret_cast<u32*>(p_d), len, 0, 0, len};
    detail::batched_invert<Field>(nullptr, &buffer_size, p_ptr, 0);
    hipMalloc(&temp_buffer, buffer_size);

    hipEvent_t start_gpu, end_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&end_gpu);
    hipEventRecord(start_gpu);

    detail::batched_invert<Field>(temp_buffer, 0, p_ptr, 0);

    hipEventRecord(end_gpu);
    hipEventSynchronize(end_gpu);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_gpu, end_gpu);
    std::cout << "gpu Time: " << milliseconds << "ms" << std::endl;
    hipMemcpy(q, p_d, len * sizeof(Field), hipMemcpyDeviceToHost);

    for (u64 i = 0; i <= len - 1; i++) {
        CHECK(q[i] == p[i].invert());
    }

    hipFree(p_d);
    hipFree(temp_buffer);

    delete[] p;
    delete[] q;
}