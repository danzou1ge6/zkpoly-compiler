#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest/doctest.h>
#include "../src/poly_basic.cuh"
#include <iostream>

using mont::u32;
using mont::u64;
using Field = bn254_fr::Element;
using Number = mont::Number<Field::LIMBS>;

u64 len = 1 << 24;

TEST_CASE("naive poly add") {
    std::cout << "testing the naive poly add" << std::endl;
    Field * a, *dst;
    a = new Field [len];
    dst = new Field [len];

    for (u64 i = 0; i < len; i++) {
        a[i] = Field::host_random();
    }

    Field *a_d;
    hipMalloc(&a_d, len * Field::LIMBS * sizeof(u32));

    u32 block = 1024;
    u32 grid = (len - 1) / block + 1;
    auto a_iter = detail::SliceIterator<Field>(a_d, len);
    detail::poly_add<Field><<<grid, block >>>(a_iter, a_iter, a_iter, len);

    hipMemcpy(a_d, a, len * Field::LIMBS * sizeof(u32), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    detail::poly_add<Field><<<grid, block >>>(a_iter, a_iter, a_iter, len);

    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "Time: " << milliseconds << "ms" << std::endl;

    hipMemcpy(dst, a_d, len * Field::LIMBS * sizeof(u32), hipMemcpyDeviceToHost);

    for (u64 i = 0; i < len; i++) {
        CHECK(dst[i] == a[i] + a[i]);
    }

    delete[] a;
    delete[] dst;
    hipFree(a_d);
}

TEST_CASE("naive poly mul") {
    std::cout << "testing the naive poly mul" << std::endl;
    Field * a, *dst;
    a = new Field [len];
    dst = new Field [len];

    for (u64 i = 0; i < len; i++) {
        a[i] = Field::host_random();
    }

    Field *a_d;
    hipMalloc(&a_d, len * Field::LIMBS * sizeof(u32));

    u32 block = 1024;
    u32 grid = (len - 1) / block + 1;

    hipMemcpy(a_d, a, len * Field::LIMBS * sizeof(u32), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    auto a_iter = detail::SliceIterator<Field>(a_d, len);

    detail::poly_mul<Field><<<grid, block >>>(a_iter, a_iter, a_iter, len);

    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "Time: " << milliseconds << "ms" << std::endl;

    hipMemcpy(dst, a_d, len * Field::LIMBS * sizeof(u32), hipMemcpyDeviceToHost);

    for (u64 i = 0; i < len; i++) {
        CHECK(dst[i] == a[i] * a[i]);
    }

    delete[] a;
    delete[] dst;
    hipFree(a_d);
}
