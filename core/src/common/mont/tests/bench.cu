#include "hip/hip_runtime.h"
#include "../src/bn254_fq.cuh"

#include <iostream>

using bn254_fq::Element;
using mont::u32;

const u32 BATCH = 1;
const u32 THREADS = 512;
const u32 ITERS = 2000;

__global__ void bench(Element *r, const Element *a)
{
  Element v = *a;
  for (u32 i = 0; i < BATCH; i++)
  {
    v = v * v;
  }
  *r = v;
}

int main()
{
  float total_time = 0;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  u32 grid_size = 32 * deviceProp.multiProcessorCount;

  for (u32 i = 0; i < ITERS; i++)
  {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    Element *r, *a;
    hipMalloc(&r, sizeof(Element));
    hipMalloc(&a, sizeof(Element));

    auto ha = Element::host_random();
    hipMemcpy(a, &ha, sizeof(Element), hipMemcpyHostToDevice);

    hipEventRecord(start);
    bench<<<grid_size, THREADS>>>(r, a);
    hipEventRecord(stop);

    auto err = hipGetLastError();
    if (err != hipSuccess)
    {
      std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
      return 1;
    }

    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    total_time += elapsed_time;
  }

  std::cout << THREADS * BATCH * ITERS * grid_size / total_time * 1000 << std::endl;

  return 0;
}
