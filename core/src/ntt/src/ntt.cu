#include "ntt.h"
#include "ssip_ntt.cuh"
#include "precompute.cuh"
#include "recompute_ntt.cuh"
#include "distribute_powers.cuh"

hipError_t ssip_ntt(PolyPtr x, const unsigned int *twiddle, unsigned int log_len, hipStream_t stream, const unsigned int max_threads_stage1_log, const unsigned int max_threads_stage2_log) {
    return detail::ssip_ntt<NTT_FIELD>(x, twiddle, log_len, stream, max_threads_stage1_log, max_threads_stage2_log);
}

hipError_t ssip_precompute(unsigned int *twiddle, unsigned int len, const unsigned int *unit) {
    detail::gen_roots_cub<NTT_FIELD> gen_roots;
    return gen_roots(twiddle, len, NTT_FIELD::load(unit));
}

hipError_t recompute_ntt(PolyPtr x, const unsigned int *pq_d, unsigned int pq_deg, const unsigned int *omegas_d, unsigned int log_len, hipStream_t stream, const unsigned int max_threads_stage1_log, const unsigned int max_threads_stage2_log) {
    return detail::recompute_ntt<NTT_FIELD>(x, pq_d, pq_deg, omegas_d, log_len, stream, max_threads_stage1_log, max_threads_stage2_log);
}

void gen_pq_omegas(unsigned int *pq, unsigned int *omegas, unsigned int pq_deg, unsigned int len, unsigned int *unit) {
    detail::gen_pq_omegas<NTT_FIELD>(pq, omegas, pq_deg, len, NTT_FIELD::load(unit));
}

hipError_t distribute_powers(PolyPtr poly, const unsigned int *powers, unsigned long long power_num, hipStream_t stream) {
    return detail::distribute_powers<NTT_FIELD>(poly, reinterpret_cast<const NTT_FIELD*>(powers), power_num, stream);
}